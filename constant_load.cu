#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cstdio>
#include <chrono>
#include <random>

// Flops = num_ops * gpu_loops * iterations * blocks * threads / time_seconds

__global__ void testKernel(float* A, float* B, float* C, long long int gpu_loops, long long int *timers) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    long long int start = clock64();
    // while ((clock64() - start) <= gpu_loops) {
    for (int i=0; i<gpu_loops; i++) {
        C[idx] = .997 * A[idx] + .998 * B[idx]; // 3 Floating Point Operations
    }
    timers[idx] += clock64() - start;
}

void print_usage(int argc, char* argv[]) {
    printf("Usage: %s [# blocks] [# threads/block] [seconds] [gpu_loops]\n", argv[0]);
}

int main(int argc, char* argv[]) {
    if (argc != 5) {
        print_usage(argc, argv);
        return 0;
    }

    int blocks = std::stoi(argv[1]);
    int threads = std::stoi(argv[2]);
    int runtime = std::stoi(argv[3]); // In seconds
    int gpu_loops = std::stoi(argv[4]); // In seconds

    if (blocks <= 0 || threads <= 0) {
        print_usage(argc, argv);
        return 0;
    }

    // Allocate memory on host and fill with random numbers
    float* hA = new float[threads*blocks];
    float* hB = new float[threads*blocks];
    float* hC = new float[threads*blocks];
    for(int i = 0; i < threads*blocks; i++) {
        hA[i] = float(std::rand())/float((RAND_MAX));
        hB[i] = float(std::rand())/float((RAND_MAX));
    }
    long long int* hTimers = new long long int[threads*blocks];

    // Allocate memory on GPU and copy data
    float *dA, *dB, *dC;
    hipMalloc(&dA, sizeof(float) * threads * blocks);
    hipMalloc(&dB, sizeof(float) * threads * blocks);
    hipMalloc(&dC, sizeof(float) * threads * blocks);
    hipMemcpy(dA, hA, sizeof(float) * threads * blocks, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, sizeof(float) * threads * blocks, hipMemcpyHostToDevice);
    long long int* dTimers;
    hipMalloc(&dTimers, sizeof(long long int) * threads * blocks);
    hipMemcpy(dTimers, hTimers, sizeof(long long int) * threads * blocks, hipMemcpyHostToDevice);

    // Initialize timing variables and start timer
    float time_ms;
    hipEvent_t gpu_start, gpu_stop;
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);
    hipEventRecord(gpu_start);

    // Run computations
    int i = 0;
    printf("Using %i blocks with %i threads/block for %i seconds.\n", blocks, threads, runtime);
    std::chrono::system_clock::time_point end = std::chrono::system_clock::now() + std::chrono::seconds(runtime);
    while (std::chrono::system_clock::now() < end) {
        testKernel<<<blocks, threads>>>(dA, dB, dC, gpu_loops, dTimers);
        hipDeviceSynchronize();
        i++;
    }

    // Calculate runtime
    hipEventRecord(gpu_stop);
    hipEventSynchronize(gpu_stop);
    hipEventElapsedTime(&time_ms, gpu_start, gpu_stop);
    printf("Actual time: %fms over %i iterations\n", time_ms, i);

    unsigned long long int flops = (3ULL * gpu_loops * i * blocks * threads) / (time_ms/1000ULL);
    printf("FLOPS: %llu\n", flops);

    hipMemcpy(hC, dC, sizeof(float) * threads * blocks, hipMemcpyDeviceToHost);
    hipMemcpy(hTimers, dTimers, sizeof(long long int) * threads * blocks, hipMemcpyDeviceToHost);

    // for(int i=0; i < 1; i++) printf("%llu\n", hTimers[i]);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    delete hA, hB, hC;
}