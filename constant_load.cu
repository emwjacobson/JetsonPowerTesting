#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblas.h>
#include <cstdio>
#include <chrono>
#include <random>
#include <iostream>

void print_usage(int argc, char* argv[]) {
    printf("Usage: %s [dim] [runtime]\n", argv[0]);
}

int main(int argc, char* argv[]) {
    if (argc != 3) {
        print_usage(argc, argv);
        return 0;
    }

    int matrix_dim = std::stoi(argv[1]);
    int runtime = std::stoi(argv[2]);

    // Allocate memory on host and fill with random numbers
    float *A, *B, *C;
    hipMallocManaged(&A, sizeof(float) * matrix_dim * matrix_dim);
    hipMallocManaged(&B, sizeof(float) * matrix_dim * matrix_dim);
    hipMallocManaged(&C, sizeof(float) * matrix_dim * matrix_dim);
    for(int i = 0; i<matrix_dim; i++) {
        A[i] = float(std::rand())/float((RAND_MAX));
        B[i] = float(std::rand())/float((RAND_MAX));
        C[i] = float(std::rand())/float((RAND_MAX));
    }

    // Initialize timing variables and start timer
    float time_ms;
    hipEvent_t gpu_start, gpu_stop;
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Run computations
    int i = 0;
    float alpha = 1, beta = 0;
    printf("Using a %i x %i matrix for %i seconds.\n", matrix_dim, matrix_dim, runtime);
    std::chrono::system_clock::time_point end = std::chrono::system_clock::now() + std::chrono::seconds(runtime);

    hipEventRecord(gpu_start);
    while (std::chrono::system_clock::now() < end) {
        hipblasSgemm(handle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_N,
                matrix_dim,
                matrix_dim,
                matrix_dim,
                &alpha, A, matrix_dim,
                B, matrix_dim,
                &beta, C, matrix_dim);
        hipDeviceSynchronize();
        i++;
    }
    hipEventRecord(gpu_stop);

    // Calculate runtime
    hipEventSynchronize(gpu_stop);
    hipEventElapsedTime(&time_ms, gpu_start, gpu_stop);
    printf("Actual time: %fms over %i iterations\n", time_ms, i);

    hipFree(A);
    hipFree(B);
    hipFree(C);
}