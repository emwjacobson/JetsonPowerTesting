#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cstdio>
#include <chrono>
#include <random>

// Flops = (iterations * num_ops_per_item * blocks * threads) / time_seconds
// (14586 * 3 * 128 * 1024 ) / 5 = 1.14x10^9 FLOPS

__global__ void testKernel(float* A, float* B, float* C, unsigned long long int *timers) {
    // Add timer here
    // https://github.com/zchee/cuda-sample/blob/master/0_Simple/clock/clock.cu

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned long long int start = clock();
    while ((clock() - start) <= 1000000) {
        C[idx] = .997 * A[idx] + .998 * B[idx]; // 3 Floating Point Operations
    }
    // timers[idx] += clock() - start;
    timers[idx]++;
}

void print_usage(int argc, char* argv[]) {
    printf("Usage: %s [# blocks] [# threads/block] [seconds]\n", argv[0]);
}

int main(int argc, char* argv[]) {
    if (argc != 4) {
        print_usage(argc, argv);
        return 0;
    }

    int blocks = std::stoi(argv[1]);
    int threads = std::stoi(argv[2]);
    int runtime = std::stoi(argv[3]); // In seconds

    if (blocks <= 0 || threads <= 0) {
        print_usage(argc, argv);
        return 0;
    }

    // Allocate memory on host and fill with random numbers
    float* hA = new float[threads*blocks];
    float* hB = new float[threads*blocks];
    float* hC = new float[threads*blocks];
    for(int i = 0; i < threads*blocks; i++) {
        hA[i] = float(std::rand())/float((RAND_MAX));
        hB[i] = float(std::rand())/float((RAND_MAX));
    }
    unsigned long long int* hTimers = new unsigned long long int[threads*blocks];

    // Allocate memory on GPU and copy data
    float *dA, *dB, *dC;
    hipMalloc(&dA, sizeof(float) * threads * blocks);
    hipMalloc(&dB, sizeof(float) * threads * blocks);
    hipMalloc(&dC, sizeof(float) * threads * blocks);
    hipMemcpy(dA, hA, sizeof(float) * threads * blocks, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, sizeof(float) * threads * blocks, hipMemcpyHostToDevice);
    unsigned long long int* dTimers;
    hipMalloc(&dTimers, sizeof(unsigned long long int) * threads * blocks);
    hipMemcpy(dTimers, hTimers, sizeof(unsigned long long int) * threads * blocks, hipMemcpyHostToDevice);

    // Initialize timing variables and start timer
    float time_ms;
    hipEvent_t gpu_start, gpu_stop;
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);
    hipEventRecord(gpu_start);

    // Run computations
    int i = 0;
    printf("Using %i blocks with %i threads/block for %i seconds.\n", blocks, threads, runtime);
    std::chrono::system_clock::time_point end = std::chrono::system_clock::now() + std::chrono::seconds(runtime);
    while (std::chrono::system_clock::now() < end) {
        testKernel<<<blocks, threads>>>(dA, dB, dC, dTimers);
        hipDeviceSynchronize();
        i++;
    }

    // Calculate runtime
    hipEventRecord(gpu_stop);
    hipEventSynchronize(gpu_stop);
    hipEventElapsedTime(&time_ms, gpu_start, gpu_stop);
    printf("Actual time: %fms over %i iterations\n", time_ms, i);

    hipMemcpy(hC, dC, sizeof(float) * threads * blocks, hipMemcpyDeviceToHost);
    hipMemcpy(hTimers, dTimers, sizeof(unsigned long long int) * threads * blocks, hipMemcpyDeviceToHost);

    for(int i=0; i < 1; i++) printf("%llu\n", hTimers[i]);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    delete hA, hB, hC;
}